#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#define N 1000

__global__ void vecadd(int a[N], int b[N], int c[N]) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

int a[N], b[N], c[N], ref[N];

TEST(Cuda, VecAdd) {
  int *cuda_a, *cuda_b, *cuda_c;
  hipMalloc(&cuda_a, sizeof a);
  hipMalloc(&cuda_b, sizeof b);
  hipMalloc(&cuda_c, sizeof c);

  for (int i = 0; i < N; ++i) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
    ref[i] = a[i] + b[i];
  }

  hipMemcpy(cuda_a, a, sizeof a, hipMemcpyHostToDevice);
  hipMemcpy(cuda_b, b, sizeof b, hipMemcpyHostToDevice);

  vecadd<<<1, 1024>>>(cuda_a, cuda_b, cuda_c);

  hipMemcpy(c, cuda_c, sizeof c, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i) {
    ASSERT_EQ(c[i], ref[i]);
  }
}

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
